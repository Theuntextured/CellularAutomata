#include "hip/hip_runtime.h"
#include "Engine.cuh"

Engine::Engine()
{
	UseBufferA = true;
	wm = new WindowManager();

	CellType* InitializationBuffer = new CellType[GridCount];
	err = hipMalloc(&CellBufferA, sizeof(CellType) * GridCount);
	ERRORCHECK;
	err = hipMalloc(&CellBufferB, sizeof(CellType) * GridCount);
	ERRORCHECK;
	InitializationBuffer = new CellType[GridCount];
	SetupInitialBuffer(InitializationBuffer);

	err = hipMemcpy(CellBufferA, InitializationBuffer, sizeof(CellType) * GridCount, hipMemcpyHostToDevice);
	ERRORCHECK;
	delete[] InitializationBuffer;
	clock.restart();
}

bool Engine::Tick()
{
	SETUPCUDAFUNCTION(ProcessCells, GridCount, !UseBufferA ? CellBufferA : CellBufferB, UseBufferA ? CellBufferA : CellBufferB);
	ERRORCHECKLAST;
	err = hipDeviceSynchronize();
	ERRORCHECK;
	wm->DrawScene(UseBufferA ? CellBufferA : CellBufferB);
	UseBufferA = !UseBufferA;
	if (!wm->Tick()) return false;
	//printf("%i\n", sf::microseconds(1000000 / ((double)MaxFPS) - clock.getElapsedTime().asMicroseconds()));
	if(1000000 / ((double)MaxFPS) - clock.getElapsedTime().asMicroseconds() > 0)sf::sleep(sf::microseconds(1000000 / ((double)MaxFPS) - clock.getElapsedTime().asMicroseconds()));
	clock.restart();
}

void Engine::SetupInitialBuffer(CellType* bufferStart)
{
	srand(static_cast <unsigned> (time(0)));
	for (int i = 0; i < GridCount; i++) {
		//bufferStart[i] = 1;
		//continue;
		bufferStart[i] = (CellType)(rand() % CellType::MAX);
	}
}

__global__ void ProcessCells(CellType* NewBuffer, CellType* OldBuffer)
{
	GETID(GridCount);

	int x = id % Width;
	int y = id / Width;

	short n[CellType::MAX];
	for (sf::Uint8 i = None; i < CellType::MAX; i++) n[i] = 0;
	

	//get neighbours
	for (int dx = -ViewDistance; dx <= ViewDistance; ++dx) {
		for (int dy = -ViewDistance; dy <= ViewDistance; ++dy) {
			if (IsValidPosition(x + dx, y + dy) && dx != 0 && dy != 0) 
				++n[OldBuffer[IndexFromPosition(x + dx, y + dy)]];
		}
	}

	NewBuffer[id] = GetNewCell(OldBuffer[id], n);
}
