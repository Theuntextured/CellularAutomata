#include "hip/hip_runtime.h"
#include "WindowManager.cuh"

WindowManager::WindowManager()
{
	Window.create(sf::VideoMode(WindowWidth, WindowHeight), sf::String("Cellular Automata"), sf::Style::None);

    err = hipMalloc(&d_PixelBuffer, sizeof(sf::Uint8) * GridCount * 4);
    ERRORCHECK;
    PixelBuffer = new sf::Uint8[GridCount * 4];

    RenderTexture.create(Width, Height);
    RenderTexture.setSmooth(false);
    RenderSprite.setScale((float)WindowWidth / (float)Width, (float)WindowHeight / (float)Height);
}

bool WindowManager::Tick()
{
    sf::Event event;
    while (Window.pollEvent(event))
    {
        if (event.type == sf::Event::Closed)
            Window.close();
    }

    return Window.isOpen();
}

void WindowManager::DrawScene(CellType* Cells)
{
    SETUPCUDAFUNCTION(RenderCells, GridCount, Cells, d_PixelBuffer);
    ERRORCHECKLAST;
    err = hipDeviceSynchronize();
    ERRORCHECK;
    err = hipMemcpy(PixelBuffer, d_PixelBuffer, sizeof(sf::Uint8) * 4 * GridCount, hipMemcpyDeviceToHost);
    RenderTexture.update(PixelBuffer);
    RenderSprite.setTexture(RenderTexture);
    Window.clear();
    Window.draw(RenderSprite);
    Window.display();
}

__global__ void RenderCells(CellType* Cells, sf::Uint8* pixels)
{
    GETID(GridCount);
    pixels[id * 4 + 0] = GetCellColor(Cells[id], r);
    pixels[id * 4 + 1] = GetCellColor(Cells[id], g);
    pixels[id * 4 + 2] = GetCellColor(Cells[id], b);
    pixels[id * 4 + 3] = GetCellColor(Cells[id], a);
}
